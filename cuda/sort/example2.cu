#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <glib-2.0/glib.h>

#define N 10
#define LEN 1000

__global__ void BubbleSort(int *vec) {

	int n = blockIdx.x;

	if (n < N) {

		int i, j, aux;
		int ini = n * LEN;
		int fin = ini + LEN;

		for (i = ini; i < fin - 1; i++) {
			for (j = i + 1; j < fin; j++) {

				if (vec[j] < vec[i]) {
					aux = vec[i];
					vec[i] = vec[j];
					vec[j] = aux;
				}
			}
		}
	}
}

int main() {

	int *arrays_cpu, *arrays_device;
	
	// Melhor performance ao utilizar uma representacao de 1 dimensão na GPU
	arrays_cpu = (int*) malloc((N * LEN) * sizeof(int));
	hipMalloc((void**) &arrays_device, (N * LEN) * sizeof(int));

	srand(time (NULL));

	// Mostra conteudo de todos N arrays pre-ordenacao
	printf("Arrays:\n");
	for (int i = 0; i < (N * LEN); i++) {
		
		if (i % LEN == 0)
			printf("\n");

		arrays_cpu[i] = rand() % LEN;
		printf("[%d] = %d\n", i, arrays_cpu[i]);
	}

	// Copia conteudo DRAM -> VRAM
	hipMemcpy(arrays_device, arrays_cpu, (N * LEN) * sizeof(int), hipMemcpyHostToDevice);

	// Inicia contagem tempo de execucao
	GTimer* timer = g_timer_new();

	// Despacha N blocos de execucao paralela na GPU
	BubbleSort<<<N, 1>>> (arrays_device);

	// Finaliza contagem de tempo
	g_timer_stop(timer);	
	gulong micro;
	double elapsed = g_timer_elapsed(timer, &micro);

	// Copia conteudo VRAM -> DRAM
	hipMemcpy(arrays_cpu, arrays_device, (N * LEN) * sizeof(int), hipMemcpyDeviceToHost);
		
	// Conteudo de todos N arrays apos a ordenacao
	printf("\nArrays Ordenados:\n");
	for (int i = 0; i < (N * LEN); i++) {
		
		if (i % LEN == 0)
			printf("\n");

		printf("[%d] = %d\n", i, arrays_cpu[i]);
	}

	printf("Tempo de Execucao: %lf segundos\n", elapsed);

	// Desaloca memória da GPU
	hipFree(arrays_device);
	free(arrays_cpu);
	return 0;
}